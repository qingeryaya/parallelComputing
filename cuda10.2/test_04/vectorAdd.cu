
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <cstdlib> // 包含头文件以使用 rand() 函数

#define LENVECTOR 10240 * 10240
#define THREADSPERBLOCK 256

using namespace std;

__global__ void vectorAdd(int *d_a, int *d_b, int *d_c)
{
        int tId = blockIdx.x * blockDim.x + threadIdx.x;
        d_c[tId] = d_a[tId] + d_b[tId];
}

int main(int argc, char const *argv[])
{
        int *a = nullptr, *b = nullptr, *c1 = nullptr, *c2 = nullptr;
        int *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

        a = (int *)malloc(sizeof(int) * LENVECTOR);
        b = (int *)malloc(sizeof(int) * LENVECTOR);
        c1 = (int *)malloc(sizeof(int) * LENVECTOR);
        c2 = (int *)malloc(sizeof(int) * LENVECTOR);

        int lower_bound = 1;
        int upper_bound = 100;

        for (int i = 0; i < LENVECTOR; i++)
        {
                a[i] = std::rand() % (upper_bound - lower_bound + 1) + lower_bound;
                b[i] = std::rand() % (upper_bound - lower_bound + 1) + lower_bound;
        }
        hipMalloc((void **)&d_a, sizeof(int) * LENVECTOR);
        hipMalloc((void **)&d_b, sizeof(int) * LENVECTOR);
        hipMalloc((void **)&d_c, sizeof(int) * LENVECTOR);
        hipMemcpy(d_a, a, sizeof(int) * LENVECTOR, hipMemcpyKind::hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, sizeof(int) * LENVECTOR, hipMemcpyKind::hipMemcpyHostToDevice);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        int THREADSPERBLOCK_TMP = 32;
        while (1)
        {
                hipEventRecord(start);
                // vectorAdd<<<LENVECTOR / THREADSPERBLOCK, THREADSPERBLOCK>>>(d_a, d_b, d_c);
                vectorAdd<<<LENVECTOR / THREADSPERBLOCK_TMP, THREADSPERBLOCK_TMP>>>(d_a, d_b, d_c);

                hipMemcpy(c1, d_c, sizeof(int) * LENVECTOR, hipMemcpyKind::hipMemcpyDeviceToHost);
                // 记录结束时间
                hipEventRecord(stop);
                // 同步 GPU
                hipEventSynchronize(stop);
                // 计算执行时间
                float milliseconds = 0.0;
                hipEventElapsedTime(&milliseconds, start, stop);
                cout << "GPU Execution Time: " << milliseconds << " ms" << "   THREADSPERBLOCK_TMP:"<<THREADSPERBLOCK_TMP<<endl;
                bool tag = true;
                for (int i = 0; i < LENVECTOR; i++)
                {
                        c2[i] = a[i] + b[i];
                        if (c2[i] != c1[i])
                        {
                                tag = false;
                        }
                }
                if (tag)
                {
                        cout << "ok" << endl;
                }
                else
                {
                        cout << "error" << endl;
                }
                THREADSPERBLOCK_TMP+=32;
                if(THREADSPERBLOCK_TMP>1024)
                {
                        break;
                }
        }

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        free(a);
        free(b);
        free(c1);
        free(c2);

        return 0;
}