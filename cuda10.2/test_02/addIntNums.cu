
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

using namespace std;

__global__ void add(int *d_a, int *d_b, int *d_c)
{
        *d_c = *d_a + *d_b;
}

int main(int argc, char const *argv[])
{

        int a = 10, b = 20, c;
        int *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

        hipMalloc((void **)&d_a, sizeof(int));
        hipMalloc((void **)&d_b, sizeof(int));
        hipMalloc((void **)&d_c, sizeof(int));

        hipMemcpy(d_a, &a, sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);
        hipMemcpy(d_b, &b, sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

        add<<<1, 1>>>(d_a, d_b, d_c);
        hipMemcpy(&c, d_c, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

        cout << a << " + " << b << " = " << c << endl;

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        return 0;
}